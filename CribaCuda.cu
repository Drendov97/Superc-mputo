#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

//Device Code
__global__ void Criba(int* Nums, int* Prims,int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int k = 0,j=0,x=0,p,N=100;
	
	if(i < tam )
	{
        for(j=2;j<=N;j++){
            if(Nums[j] != 1 || j == 2)
            {
                Prims[x]=j;
                printf("El número %d es primo",j);
                for(p=2;(p*j)<=n;p++){
                    Nums[(p*j)]=1;
                }
                x++;

            }
        }
        printf("Hay %d números primos",x);
		
	}	
}


int main()
{
	int tam = 1000, i = 0, j = 0;
	size_t size = tam * tam * sizeof(float);
	
	// Allocate input vectors h_A and h_B in host memory
	float* h_A = (float*) malloc (size);
	float* h_B = (float*) malloc (size);
	float* h_C = (float*) malloc (size);

	//Initialize input Vectors
	for(i=0; i < tam; i++)
	{
		for(j=0; j < tam; j++)
		{
			srand(time(NULL));
			*(h_A + (i * tam) + j) = drand48() * (10.0 - 0.0) + 0.0;
			*(h_B + (i * tam) + j) = drand48() * (10.0 - 0.0) + 0.0;
		}
	}

	//Allocate vectors in device memory
	float* d_A;
	hipMalloc(&d_A, size);
	float* d_B;
	hipMalloc(&d_B, size);
	float* d_C;
	hipMalloc(&d_C, size);

	// Copy vectors from host memory to device memory
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

	// Invoke kernel
	dim3 dimGrid(256,256);
	dim3 dimBlock(tam,tam);

	VecAdd<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, tam);
	
	// Copy result from device memory to hostmemory
	// h_C contains the result in host memory
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	//impresion A
/*	for(i = 0; i<tam; i++)
	{
		for(j=0; j<tam; j++)
		{			
			printf("\tA[%d][%d]= %f", i, j, *(h_A + (i * tam) + j) );			
		}
		printf("\n");
	}
	printf("\n");
	//impresion B
	for(i = 0; i<tam; i++)
	{
		for(j=0; j<tam; j++)
		{			
			printf("\tB[%d][%d]= %f",i, j, *(h_B + (i * tam) + j) );			
		}
		printf("\n");
	}
	printf("\n");
	//impresion C
	for(i = 0; i<tam; i++)
	{
		for(j=0; j<tam; j++)
		{			
			printf("\tC[%d][%d]= %f",i, j, *(h_C + (i * tam) + j) );			
		}
		printf("\n");
	}
	printf("\n");*/

	//Free host memory
	free(h_A);
	free(h_B);
	free(h_C);
}
